#include "hip/hip_runtime.h"
#include "gtest/gtest.h"

#include "cudex/memory.cu.h"
#include "cudex/launcher.cu.h"
#include "cudex/device_utils.cu.h"

using namespace cudex;

namespace
{

__global__ void setData(DeviceSpan<int> span)
{
    const size_t index = threadLinearIndex();
    if (index >= span.size())
    {
        return;
    }

    span[index] += 2;
}

}

TEST(launcher, run_1d)
{
    constexpr size_t SIZE = 1e6;
    HostDeviceMemory<int> mem(SIZE);

    EXPECT_EQ(mem.size(), SIZE);

    int cnt = 0;
    for (int& v : mem.host())
    {
        v = cnt ++;
    } 

    EXPECT_EQ(cnt, SIZE);

    mem.copyHostToDeviceAsync();

    auto launcher = Launcher().async().size1D(SIZE);
    launcher.run(setData, mem.device());

    mem.copyDeviceToHost();

    cnt = 0;
    for (const auto& v: mem.host())
    {
        EXPECT_EQ(v, cnt++ + 2);
    }

    EXPECT_EQ(cnt, SIZE);
}
